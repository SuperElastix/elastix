#include "hip/hip_runtime.h"
/*======================================================================

  This file is part of the elastix software.

  Copyright (c) University Medical Center Utrecht. All rights reserved.
  See src/CopyrightElastix.txt or http://elastix.isi.uu.nl/legal.php for
  details.

     This software is distributed WITHOUT ANY WARRANTY; without even
     the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
     PURPOSE. See the above copyright notices for more information.

======================================================================*/
#include "cudaResampleImageFilter.cuh"
#include "CI/cubicPrefilter3D.cu"
#include "cudaInlineFunctions.h"


__constant__ float3 CUInputImageSpacing;
__constant__ float3 CUInputImageOrigin;
__constant__ float3 CUOutputImageSpacing;
__constant__ float3 CUOutputImageOrigin;
__constant__ float3 CUGridSpacing;
__constant__ float3 CUGridOrigin;
__constant__ int3   CUGridSize;
__constant__ float  CUDefaultPixelValue;


#include "cudaDeformationsKernel.cu"


/**
 * ******************* Constructor ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::CUDAResampleImageFilter()
  : m_CoeffsX( NULL )
  , m_CoeffsY( NULL )
  , m_CoeffsZ( NULL )
  , m_InputImage( NULL )
  , m_InputImageSize( make_uint3( 0, 0, 0 ) )
  , m_Device( 0 )
  , m_MaxNumberOfVoxelsPerIteration( 1 << 20 )
{
  this->m_CastOnGPU = false;
  this->m_UseFastCUDAKernel = false;

} // end Constructor


/**
 * ******************* Destructor ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::~CUDAResampleImageFilter()
{
  this->cudaUnInit();
} // end Destructor


/**
 * ******************* cudaInit ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaInit( void )
{
  this->checkExecutionParameters();
  cuda::hipSetDevice( this->m_Device ); // always 0?
  this->m_ChannelDescCoeff = hipCreateChannelDesc<TInternalImageType>();

} // end cudaInit()


/**
 * ******************* cudaUnInit ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaUnInit( void )
{
  cuda::hipUnbindTexture( m_tex_coeffsX );
  cuda::hipUnbindTexture( m_tex_coeffsY );
  cuda::hipUnbindTexture( m_tex_coeffsZ );
  cuda::hipUnbindTexture( m_tex_inputImage );
  cuda::hipFreeArray( this->m_CoeffsX );
  cuda::hipFreeArray( this->m_CoeffsY );
  cuda::hipFreeArray( this->m_CoeffsZ );
  cuda::hipFreeArray( this->m_InputImage );
  cuda::hipFree( this->m_OutputImage );

} // end cudaUnInit()


/**
 * ******************* checkExecutionParameters ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
int
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::checkExecutionParameters( void )
{
  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount( &deviceCount );
  return ( err == hipSuccess ) ? ( deviceCount == 0 ) : 1;

} // end checkExecutionParameters()


/**
 * ******************* cudaCopyImageSymbols ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaCopyImageSymbols(
  const float3 & inputImageSpacing,  const float3 & inputImageOrigin,
  const float3 & outputImageSpacing, const float3 & outputImageOrigin,
  const float defaultPixelValue )
{
  /* Copy some constant parameters to the GPU's constant cache. */
  cuda::hipMemcpyToSymbol(HIP_SYMBOL( CUInputImageSpacing),  inputImageSpacing,
    hipMemcpyHostToDevice );
  cuda::hipMemcpyToSymbol(HIP_SYMBOL( CUInputImageOrigin),   inputImageOrigin,
    hipMemcpyHostToDevice );
  cuda::hipMemcpyToSymbol(HIP_SYMBOL( CUOutputImageSpacing), outputImageSpacing,
    hipMemcpyHostToDevice );
  cuda::hipMemcpyToSymbol(HIP_SYMBOL( CUOutputImageOrigin),  outputImageOrigin,
    hipMemcpyHostToDevice );
  cuda::hipMemcpyToSymbol(HIP_SYMBOL( CUDefaultPixelValue),  defaultPixelValue,
    hipMemcpyHostToDevice );

} // end cudaCopyImageSymbols()


/**
 * ******************* cudaCopyGridSymbols ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaCopyGridSymbols( const float3 & gridSpacing,
  const float3 & gridOrigin, const uint3 & gridSize )
{
  /* Copy some constant parameters to the GPU's constant cache. */
  cuda::hipMemcpyToSymbol(HIP_SYMBOL( CUGridSpacing), gridSpacing, hipMemcpyHostToDevice );
  cuda::hipMemcpyToSymbol(HIP_SYMBOL( CUGridOrigin),  gridOrigin,  hipMemcpyHostToDevice );
  cuda::hipMemcpyToSymbol(HIP_SYMBOL( CUGridSize),    gridSize,    hipMemcpyHostToDevice );

} // end cudaCopyGridSymbols()


/**
 * ******************* cudaMallocTransformationData ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaMallocTransformationData( const uint3 & gridSize,
  const TInterpolatorPrecisionType* params )
{
  const unsigned int nrOfParametersPerDimension = gridSize.x * gridSize.y * gridSize.z;
  hipExtent gridExtent = make_hipExtent( gridSize.x, gridSize.y, gridSize.z );

  /* Allocate memory on the GPU for the interpolation texture. */
  cuda::hipMalloc3DArray( &this->m_CoeffsX, &this->m_ChannelDescCoeff, gridExtent );
  cuda::hipMalloc3DArray( &this->m_CoeffsY, &this->m_ChannelDescCoeff, gridExtent );
  cuda::hipMalloc3DArray( &this->m_CoeffsZ, &this->m_ChannelDescCoeff, gridExtent );

  /* Convert TInterpolatorPrecisionType to float, only thing textures support. */
#if 1
  //clock_t start = clock();
  TInternalImageType* params_tmp = new TInternalImageType[ nrOfParametersPerDimension * 3 ];
  for ( size_t i = 0; i != nrOfParametersPerDimension * 3; ++i )
  {
    params_tmp[ i ] = static_cast<TInternalImageType>( params[ i ] );
  }
  //std::cout << "parameter type conversion took "
  //  << clock() - start << "ms for "
  //  << nrOfParametersPerDimension * 3 << " elements" << std::endl;
  hipBindTextureToArray( m_CoeffsX, &params_tmp[ 0 * nrOfParametersPerDimension ],
    gridExtent, m_tex_coeffsX, this->m_ChannelDescCoeff );
  hipBindTextureToArray( m_CoeffsY, &params_tmp[ 1 * nrOfParametersPerDimension ],
    gridExtent, m_tex_coeffsY, this->m_ChannelDescCoeff );
  hipBindTextureToArray( m_CoeffsZ, &params_tmp[ 2 * nrOfParametersPerDimension ],
    gridExtent, m_tex_coeffsZ, this->m_ChannelDescCoeff );
  delete[] params_tmp;
#else
  /* There are some problems with Device2Device copy when src is not a pitched or 3D array. */
  TInternalImageType* params_gpu
    = cuda::hipMalloc<TInternalImageType>( nrOfParametersPerDimension );

  /* Create the B-spline coefficients texture. */
  cudaCastToType<TInterpolatorPrecisionType, TInternalImageType>(
    gridExtent, &params[ 0 * nrOfParametersPerDimension ],
    params_gpu, hipMemcpyHostToDevice, m_CastOnGPU );
  hipBindTextureToArray( m_CoeffsX, params_gpu, gridExtent, m_tex_coeffsX,
    m_ChannelDescCoeff, false, true );

  cudaCastToType<TInterpolatorPrecisionType, TInternalImageType>(
    gridExtent, &params[ 1 * nrOfParametersPerDimension ],
    params_gpu, hipMemcpyHostToDevice, m_CastOnGPU );
  hipBindTextureToArray( m_CoeffsY, params_gpu, gridExtent, m_tex_coeffsY,
    m_ChannelDescCoeff, false, true );

  cudaCastToType<TInterpolatorPrecisionType, TInternalImageType>(
    gridExtent, &params[ 2 * nrOfParametersPerDimension ],
    params_gpu, hipMemcpyHostToDevice, m_CastOnGPU );
  hipBindTextureToArray( m_CoeffsZ, params_gpu, gridExtent, m_tex_coeffsZ,
    m_ChannelDescCoeff, false, true );

  cuda::hipFree( params_gpu );
#endif

} // end cudaMallocTransformationData()


/**
 * ******************* cudaMallocImageData ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaMallocImageData( const uint3 & inputSize,
  const uint3 & outputSize, const TImageType* data )
{
  this->m_InputImageSize        = inputSize;
  this->m_OutputImageSize       = outputSize;
  this->m_NumberOfInputVoxels   = this->m_InputImageSize.x
    * this->m_InputImageSize.y * this->m_InputImageSize.z;
  size_t nrOfOutputVoxels       = this->m_OutputImageSize.x
    * this->m_OutputImageSize.y * this->m_OutputImageSize.z;
  this->m_MaxNumberOfVoxelsPerIteration = std::min(
    static_cast<unsigned int>( nrOfOutputVoxels ),
    this->m_MaxNumberOfVoxelsPerIteration );

  hipExtent volumeExtent = make_hipExtent(
    this->m_InputImageSize.x, this->m_InputImageSize.y, this->m_InputImageSize.z );

  /* Allocate in memory and PreFilter image. We need to cast to float if not
   * already, because linear filtering only works with floating point values.
   * NOTE: the input image needs to be allocated on the GPU entirely,
   * which may fail for large images and low-end GPU's.
   */
  TInternalImageType* inputImage
    = cuda::hipMalloc<TInternalImageType>( this->m_NumberOfInputVoxels );
  cudaCastToDevice( this->m_InputImageSize, data, inputImage );
  /** Prefiltering is performed in-place. */
  CubicBSplinePrefilter3D( inputImage,
    volumeExtent.width, volumeExtent.height, volumeExtent.depth );

  /* XXX - hipMemcpy3D fails if a DeviceToDevice copy src is not allocated
   * with hipMallocPitch or hipMalloc3D, so we need this hack to get the data there.
   */
  TInternalImageType* tmpImage
    = new TInternalImageType[ this->m_NumberOfInputVoxels ];
  cuda::hipMemcpy( tmpImage, inputImage,
    this->m_NumberOfInputVoxels, hipMemcpyDeviceToHost );
  cuda::hipFree( inputImage );

  /* Create the image interpolation texture. */
  cuda::hipMalloc3DArray( &this->m_InputImage, &this->m_ChannelDescCoeff, volumeExtent );
  hipBindTextureToArray( this->m_InputImage, tmpImage,
    volumeExtent, m_tex_inputImage, this->m_ChannelDescCoeff );
  delete[] tmpImage;

  /* Allocate destination array. */
  this->m_OutputImage = cuda::hipMalloc<TInternalImageType>(
    this->m_MaxNumberOfVoxelsPerIteration );

} // end cudaMallocImageData()


/**
 * ******************* GenerateData ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::GenerateData( TImageType* dst )
{
  /* Split up applying the transformation due to memory constraints and make
   * sure we never overflow the output image dimensions.
   */
  const size_t nrOfOutputVoxels = this->m_OutputImageSize.x
    * this->m_OutputImageSize.y * this->m_OutputImageSize.z;
  dim3 dimBlock( 256 );
  dim3 dimGrid( this->m_MaxNumberOfVoxelsPerIteration / dimBlock.x );
  size_t offset = 0;

  TInternalImageType* tmp_src = new TInternalImageType[ this->m_MaxNumberOfVoxelsPerIteration ];
  if ( nrOfOutputVoxels > this->m_MaxNumberOfVoxelsPerIteration )
  {
    /* Do a full run of m_MaxnrOfVoxelsPerIteration voxels. */
    for ( offset = 0; offset <= nrOfOutputVoxels - this->m_MaxNumberOfVoxelsPerIteration;
      offset += this->m_MaxNumberOfVoxelsPerIteration )
    {
      resample_image<<<dimGrid, dimBlock>>>( this->m_OutputImage,
        this->m_InputImageSize, this->m_OutputImageSize, offset, this->m_UseFastCUDAKernel );
      cuda::cudaCheckMsg( "kernel launch failed: resample_image" );
      cudaCastToHost( this->m_MaxNumberOfVoxelsPerIteration,
        this->m_OutputImage, tmp_src, &dst[ offset ] );
    }
  }

  /* Do the remainder ensuring again dimGrid * dimBlock is less than image size. */
  dimGrid = dim3((unsigned int)(nrOfOutputVoxels - offset)) / dimBlock;
  resample_image<<<dimGrid, dimBlock>>>( this->m_OutputImage,
    this->m_InputImageSize, this->m_OutputImageSize, offset, this->m_UseFastCUDAKernel );
  cuda::cudaCheckMsg( "kernel launch failed: resample_image" );
  cudaCastToHost( dimGrid.x * dimBlock.x, m_OutputImage, tmp_src, &dst[ offset ] );

  /* Do the final amount of voxels < dimBlock. */
  offset += dimGrid.x * dimBlock.x;
  dimBlock = dim3((unsigned int)(nrOfOutputVoxels - offset));
  dimGrid  = dim3( 1 );

  if ( dimBlock.x > 0 )
  {
    resample_image<<<dimGrid, dimBlock>>>( this->m_OutputImage,
      this->m_InputImageSize, this->m_OutputImageSize, offset, this->m_UseFastCUDAKernel );
    cuda::cudaCheckMsg( "kernel launch failed: resample_image" );
    cudaCastToHost( dimGrid.x * dimBlock.x, m_OutputImage, tmp_src, &dst[ offset ] );
  }
  delete[] tmp_src;

} // end GenerateData()


/**
 * ******************* hipBindTextureToArray ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
template <typename TTextureType>
hipError_t
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::hipBindTextureToArray( hipArray* dst, const TInternalImageType* src,
  const hipExtent & extent, TTextureType& tex, hipChannelFormatDesc& desc,
  bool normalized, bool onDevice )
{
  hipMemcpy3DParms copyParams = {0};
  copyParams.extent   = extent;
  copyParams.kind   = onDevice ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
  copyParams.dstArray = dst;
  copyParams.srcPtr   = make_hipPitchedPtr(
    const_cast<TInternalImageType*>(src),
    extent.width * sizeof(TInternalImageType), extent.width, extent.height );
  cuda::hipMemcpy3D( &copyParams );

  tex.normalized   = normalized;
  tex.filterMode   = cudaFilterMode;
  tex.addressMode[0] = tex.normalized ? hipAddressModeMirror: hipAddressModeClamp;
  tex.addressMode[1] = tex.normalized ? hipAddressModeMirror: hipAddressModeClamp;
  tex.addressMode[2] = tex.normalized ? hipAddressModeMirror: hipAddressModeClamp;
  return cuda::hipBindTextureToArray( tex, dst, desc );

} // end hipBindTextureToArray()


/**
 * ******************* cudaCastToHost ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaCastToHost( const size_t sizevalue, const TInternalImageType* src,
  TInternalImageType* tmp_src, TImageType* dst )
{
  cuda::hipMemcpy( tmp_src, src, sizevalue, hipMemcpyDeviceToHost );
  for ( size_t i = 0; i != sizevalue; ++i )
  {
    dst[ i ] = static_cast<TImageType>( tmp_src[i] );
  }

} // end cudaCastToHost()


/**
 * ******************* cudaCastToHost ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaCastToHost( const uint3 & sizevalue, const TInternalImageType* src, TImageType* dst )
{
  hipExtent volumeExtent = make_hipExtent( sizevalue.x, sizevalue.y, sizevalue.z );
  cudaCastToType<TInternalImageType, TImageType>(
    volumeExtent, src, dst, hipMemcpyDeviceToHost, m_CastOnGPU );

} // end cudaCastToHost()


/**
 * ******************* cudaCastToDevice ***********************
 */

template <typename TInterpolatorPrecisionType, typename TImageType, typename TInternalImageType>
void
cuda::CUDAResampleImageFilter<TInterpolatorPrecisionType, TImageType, TInternalImageType>
::cudaCastToDevice( const uint3 & sizevalue, const TImageType* src, TInternalImageType* dst )
{
  hipExtent volumeExtent = make_hipExtent( sizevalue.x, sizevalue.y, sizevalue.z );
  cudaCastToType<TImageType, TInternalImageType>(
    volumeExtent, src, dst, hipMemcpyHostToDevice, m_CastOnGPU );

} // end cudaCastToDevice()


/**
 * ******************* is_double ***********************
 */

template <class T> inline bool is_double();
template <class T> inline bool is_double() {return false;}
template <       > inline bool is_double<double>() {return true;}


/**
 * ******************* cudaCastToType ***********************
 */

template <>
float* cuda::cudaCastToType<float, float>( const hipExtent & volumeExtent,
  const float* src, float* dst, hipMemcpyKind direction, const bool useGPU )
{
  const size_t voxelsPerSlice = volumeExtent.width * volumeExtent.height;
  cuda::hipMemcpy( dst, src, voxelsPerSlice * volumeExtent.depth, direction );
  return dst;

} // end cudaCastToType()


/**
 * ******************* cudaCastToType ***********************
 */

template <class TInputImageType, class TOutputImageType>
TOutputImageType* cuda
::cudaCastToType( const hipExtent & volumeExtent,
  const TInputImageType* src, TOutputImageType* dst,
  hipMemcpyKind direction, bool useGPU )
{
  hipDeviceProp_t prop;
  size_t offset = 0;
  const size_t voxelsPerSlice = volumeExtent.width * volumeExtent.height;

  // std::max( size_t, size_t ) does not exist
  dim3 dimBlock( std::min( static_cast<int>(
    std::max( (long long)volumeExtent.width, (long long)volumeExtent.height ) ), 512 ) );
  dim3 dimGrid( (unsigned int)( voxelsPerSlice / dimBlock.x ) );

  /* Not a perfect fit, fix it */
  if ( dimBlock.x * dimGrid.x != voxelsPerSlice ) ++dimGrid.x;

  //clock_t start = clock();

  /* only devices from compute capability 1.3 support double precision on the device */
  cuda::hipGetDeviceProperties( &prop, 0 );
  bool device_less_2_0 = ( prop.major == 1 && prop.minor < 3 );

  switch ( direction )
  {
  case hipMemcpyHostToDevice:
    if ( is_double<TOutputImageType>() && device_less_2_0 )
    {
      throw std::string( "GPU doesn't support double-precision" );
    }

    if ( !useGPU )
    {
      size_t nof_elements = voxelsPerSlice * volumeExtent.depth;

      /* Allocate memory on host, copy over data (and cast) and copy results to GPU. */
      TOutputImageType* tmp = new TOutputImageType[ nof_elements ];
      for ( size_t i = 0; i != nof_elements; ++i )
      {
        tmp[ i ] = static_cast<TOutputImageType>( src[ i ] );
      }
      cuda::hipMemcpy( dst, tmp, nof_elements, hipMemcpyHostToDevice );
    }
    else
    {
      TInputImageType* tmp = cuda::hipMalloc<TInputImageType>( voxelsPerSlice );

      /* Process each slice separately, copy source to GPU, and cast/copy in kernel. */
      for ( unsigned int slice = 0; slice != volumeExtent.depth; ++slice, offset += voxelsPerSlice )
      {
        cuda::hipMemcpy( tmp, src + offset, voxelsPerSlice, hipMemcpyHostToDevice );
        cast_to_type<TInputImageType, TOutputImageType><<<dimGrid, dimBlock>>>(
          dst + offset, tmp, voxelsPerSlice );
        cuda::cudaCheckMsg( "kernel launch failed: cast_to_type" );
      }
      hipFree( tmp );
    }
    break;
  case hipMemcpyDeviceToHost:
    if ( is_double<TInputImageType>() && device_less_2_0 )
    {
      throw std::string( "GPU doesn't support double-precision" );
    }

    if ( !useGPU )
    {
      size_t nof_elements = voxelsPerSlice * volumeExtent.depth;

      /* Allocate memory on host, copy data from GPU and cast. */
      TInputImageType* tmp = new TInputImageType[ nof_elements ];
      cuda::hipMemcpy( tmp, src, nof_elements, hipMemcpyDeviceToHost );
      for ( size_t i = 0; i != nof_elements; ++i )
      {
        dst[ i ] = static_cast<TOutputImageType>( tmp[ i ] );
      }
    }
    else
    {
      TOutputImageType* tmp = cuda::hipMalloc<TOutputImageType>( voxelsPerSlice );

      /* Process each slice separately, cast/copy in kernel and copy results to host. */
      for ( unsigned int slice = 0; slice != volumeExtent.depth; ++slice, offset += voxelsPerSlice )
      {
        cast_to_type<TInputImageType, TOutputImageType><<<dimGrid, dimBlock>>>(
          tmp, src + offset, voxelsPerSlice );
        cuda::cudaCheckMsg( "kernel launch failed: cast_to_type" );
        cuda::hipMemcpy( dst + offset, tmp, voxelsPerSlice, hipMemcpyDeviceToHost );
      }
      hipFree( tmp );
    }
    break;
  case hipMemcpyHostToHost:
    break;
  case hipMemcpyDeviceToDevice:
    break;
  case hipMemcpyDefault:
    break;
  }

  return dst;

} // end cudaCastToType()


/** Template linker errors...
 * http://www.parashift.com/c++-faq-lite/templates.html#faq-35.14
 * Note that gcc requires these lines at the bottom of this file.
 */
template class cuda::CUDAResampleImageFilter<double, short, float>;
template class cuda::CUDAResampleImageFilter<double, int  , float>;
template class cuda::CUDAResampleImageFilter<double, float, float>;

