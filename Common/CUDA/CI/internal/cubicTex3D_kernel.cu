#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2010, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.
\*--------------------------------------------------------------------------*/

//! Tricubic interpolated texture lookup, using unnormalized coordinates.
//! Fast implementation, using 8 trilinear lookups.
//! @param tex  3D texture
//! @param coord  unnormalized 3D texture coordinate
template<class floatN, class T, enum hipTextureReadMode mode>
__device__ floatN CUBICTEX3D(texture<T, 3, mode> tex, float3 coord)
{
  // shift the coordinate from [0,extent] to [-0.5, extent-0.5]
  const float3 coord_grid = coord - 0.5f;
  const float3 index = floor(coord_grid);
  const float3 fraction = coord_grid - index;
  float3 w0, w1, w2, w3;
  WEIGHTS(fraction, w0, w1, w2, w3);

  const float3 g0 = w0 + w1;
  const float3 g1 = w2 + w3;
  const float3 h0 = (w1 / g0) - 0.5f + index;  //h0 = w1/g0 - 1, move from [-0.5, extent-0.5] to [0, extent]
  const float3 h1 = (w3 / g1) + 1.5f + index;  //h1 = w3/g1 + 1, move from [-0.5, extent-0.5] to [0, extent]

  // fetch the eight linear interpolations
  // weighting and fetching is interleaved for performance and stability reasons
  floatN tex000 = tex3D(tex, h0.x, h0.y, h0.z);
  floatN tex100 = tex3D(tex, h1.x, h0.y, h0.z);
  tex000 = g0.x * tex000 + g1.x * tex100;  //weigh along the x-direction
  floatN tex010 = tex3D(tex, h0.x, h1.y, h0.z);
  floatN tex110 = tex3D(tex, h1.x, h1.y, h0.z);
  tex010 = g0.x * tex010 + g1.x * tex110;  //weigh along the x-direction
  tex000 = g0.y * tex000 + g1.y * tex010;  //weigh along the y-direction
  floatN tex001 = tex3D(tex, h0.x, h0.y, h1.z);
  floatN tex101 = tex3D(tex, h1.x, h0.y, h1.z);
  tex001 = g0.x * tex001 + g1.x * tex101;  //weigh along the x-direction
  floatN tex011 = tex3D(tex, h0.x, h1.y, h1.z);
  floatN tex111 = tex3D(tex, h1.x, h1.y, h1.z);
  tex011 = g0.x * tex011 + g1.x * tex111;  //weigh along the x-direction
  tex001 = g0.y * tex001 + g1.y * tex011;  //weigh along the y-direction

  return (g0.z * tex000 + g1.z * tex001);  //weigh along the z-direction
}


// Specializations

// These specializations fill in the floatN and T class types and therefore
// allow the cubicTex3D function to be called without any template arguments,
// thus with any <> brackets.

// 1-dimensional voxels
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<float, 3, mode> tex, float3 coord) {return CUBICTEX3D<float, float, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<uchar, 3, mode> tex, float3 coord) {return CUBICTEX3D<float, uchar, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<char, 3, mode> tex, float3 coord) {return CUBICTEX3D<float, char, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<ushort, 3, mode> tex, float3 coord) {return CUBICTEX3D<float, ushort, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<short, 3, mode> tex, float3 coord) {return CUBICTEX3D<float, short, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<uint, 3, mode> tex, float3 coord) {return CUBICTEX3D<float, uint, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<int, 3, mode> tex, float3 coord) {return CUBICTEX3D<float, int, mode>(tex, coord);}
// 2-dimensional voxels
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<float2, 3, mode> tex, float3 coord) {return CUBICTEX3D<float2, float2, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<uchar2, 3, mode> tex, float3 coord) {return CUBICTEX3D<float2, uchar2, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<char2, 3, mode> tex, float3 coord) {return CUBICTEX3D<float2, char2, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<ushort2, 3, mode> tex, float3 coord) {return CUBICTEX3D<float2, ushort2, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<short2, 3, mode> tex, float3 coord) {return CUBICTEX3D<float2, short2, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<uint2, 3, mode> tex, float3 coord) {return CUBICTEX3D<float2, uint2, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<int2, 3, mode> tex, float3 coord) {return CUBICTEX3D<float2, int2, mode>(tex, coord);}
// 3-dimensional voxels
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<float3, 3, mode> tex, float3 coord) {return CUBICTEX3D<float3, float3, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<uchar3, 3, mode> tex, float3 coord) {return CUBICTEX3D<float3, uchar3, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<char3, 3, mode> tex, float3 coord) {return CUBICTEX3D<float3, char3, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<ushort3, 3, mode> tex, float3 coord) {return CUBICTEX3D<float3, ushort3, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<short3, 3, mode> tex, float3 coord) {return CUBICTEX3D<float3, short3, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<uint3, 3, mode> tex, float3 coord) {return CUBICTEX3D<float3, uint3, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<int3, 3, mode> tex, float3 coord) {return CUBICTEX3D<float3, int3, mode>(tex, coord);}
// 4-dimensional voxels
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<float4, 3, mode> tex, float3 coord) {return CUBICTEX3D<float4, float4, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<uchar4, 3, mode> tex, float3 coord) {return CUBICTEX3D<float4, uchar4, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<char4, 3, mode> tex, float3 coord) {return CUBICTEX3D<float4, char4, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<ushort4, 3, mode> tex, float3 coord) {return CUBICTEX3D<float4, ushort4, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<short4, 3, mode> tex, float3 coord) {return CUBICTEX3D<float4, short4, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<uint4, 3, mode> tex, float3 coord) {return CUBICTEX3D<float4, uint4, mode>(tex, coord);}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<int4, 3, mode> tex, float3 coord) {return CUBICTEX3D<float4, int4, mode>(tex, coord);}

// Function definitions with (float x, float y, float z) instead of (float3 coord)
// 1-dimensional voxels
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<float, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float, float, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<uchar, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float, uchar, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<char, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float, char, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<ushort, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float, ushort, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<short, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float, short, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<uint, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float, uint, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float CUBICTEX3D(texture<int, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float, int, mode>(tex, make_float3(x,y,z));}
// 2-dimensional voxels
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<float2, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float2, float2, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<uchar2, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float2, uchar2, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<char2, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float2, char2, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<ushort2, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float2, ushort2, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<short2, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float2, short2, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<uint2, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float2, uint2, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float2 CUBICTEX3D(texture<int2, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float2, int2, mode>(tex, make_float3(x,y,z));}
// 3-dimensional voxels
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<float3, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float3, float3, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<uchar3, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float3, uchar3, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<char3, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float3, char3, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<ushort3, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float3, ushort3, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<short3, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float3, short3, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<uint3, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float3, uint3, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float3 CUBICTEX3D(texture<int3, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float3, int3, mode>(tex, make_float3(x,y,z));}
// 4-dimensional voxels
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<float4, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float4, float4, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<uchar4, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float4, uchar4, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<char4, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float4, char4, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<ushort4, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float4, ushort4, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<short4, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float4, short4, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<uint4, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float4, uint4, mode>(tex, make_float3(x,y,z));}
template<enum hipTextureReadMode mode> __device__ float4 CUBICTEX3D(texture<int4, 3, mode> tex, float x, float y, float z) {return CUBICTEX3D<float4, int4, mode>(tex, make_float3(x,y,z));}

