#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2009, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.
\*--------------------------------------------------------------------------*/

#ifndef _MATH_FUNC_CUDA_H_
#define _MATH_FUNC_CUDA_H_

#include "version.cu"

typedef unsigned int uint;
typedef unsigned short ushort;
typedef unsigned char uchar;
typedef signed char schar;

inline __device__ __host__ uint UMIN(uint a, uint b)
{
  return a < b ? a : b;
}

inline __device__ __host__ uint PowTwoDivider(uint n)
{
  if (n == 0) return 0;
  uint divider = 1;
  while ((n & divider) == 0) divider <<= 1;
  return divider;
}

inline __host__ __device__ float2 operator-(float a, float2 b)
{
  return make_float2(a - b.x, a - b.y);
}

inline __host__ __device__ float3 operator-(float a, float3 b)
{
  return make_float3(a - b.x, a - b.y, a - b.z);
}

#endif  //_MATH_FUNC_CUDA_H_
